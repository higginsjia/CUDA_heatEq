#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "utils.h"
#include "dev_matrix.h"

// DIVIDE_INTO(x/y) for integers, used to determine # of blocks/warps etc.
#define DIVIDE_INTO(x,y) (((x) + (y) - 1)/(y))
// I2D to index into a linear memory space from a 2D array index pair
#define I2D(Nx, i, j) ((i) + (Nx)*(j))

// Block size in the i and j directions
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

// kernel to update temperatures - CPU version
void heat2d_cpu(int Nx, int Ny, double alp, double *in, double *out) {
	int i, j, P, W, E, S, N;
	double d2tdx2, d2tdy2;
	// loop over all points in domain (not boundary points)
	for (j = 1; j < Ny-1; j++) {
		for (i = 1; i < Nx-1; i++) {
			// find indices into linear memory for central point and neighbours
			P = I2D(Nx, i, j);
			W = I2D(Nx, i-1, j); E = I2D(Nx, i+1, j);
			S = I2D(Nx, i, j-1); N = I2D(Nx, i, j+1);

			d2tdx2 = in[W] - 2.0*in[P] + in[E];
			d2tdy2 = in[N] - 2.0*in[P] + in[S];

			out[P] = in[P] + alp*(d2tdx2 + d2tdy2);
		}
	}
}

// kernel to update temperatures - GPU version (not using shared mem)
__global__ void heat2d_gpu(int Nx, int Ny, double alp, double *in, double *out) 
{
	int i, j, P, W, E, S, N;
	double d2tdx2, d2tdy2;
	// find i and j indices of this thread
	i = blockIdx.x*(BLOCK_SIZE_X) + threadIdx.x;
	j = blockIdx.y*(BLOCK_SIZE_Y) + threadIdx.y;

	// find indices into linear memory 
	P = I2D(Nx, i, j);
	W = I2D(Nx, i-1, j); E = I2D(Nx, i+1, j);
	S = I2D(Nx, i, j-1); N = I2D(Nx, i, j+1);

	// check that thread is within domain (not on boundary or outside domain)
	if (i > 0 && i < Nx-1 && j > 0 && j < Ny-1) {
		d2tdx2 = in[W] - 2.0*in[P] + in[E];
		d2tdy2 = in[N] - 2.0*in[P] + in[S];

		out[P] = in[P] + alp*(d2tdx2 + d2tdy2);
	}
}

void heat2d_exc(double * out, double * in1, double * in2,
	const double T, const int Nx, const int Ny)
{		
	int i, j;

	for (j = 1; j < Ny-1; j++) {
		for (i = 1; i < Nx-1; i++) {
			out[I2D(Nx, i, j)] = sin(M_PI*in1[i])*sin(M_PI*in2[j]) * exp(-2.0*M_PI*M_PI*T);
		}
	}
}


int main() 
{
	int Nx, Ny, Nt;
	double alpha, *x, *y, *u_h, *oldu_h, *tmp_h, *exc;
	double * u_d;
	int i, j, iter;
	double h, T;
	double errGPU = 0.0, errCPU = 0.0;
	dim3 numBlocks, threadsPerBlock;
	double clock_h, clock_d;
	FILE *fp;

	// domain size and number of timesteps (iterations)
	Nx = 1024;
	Ny = Nx;
	Nt = 100;
	alpha = 0.25;
	h = 1.0 / (double)(Nx-1);
	T = Nt*alpha*h*h;

	// allocate temperature array on host
	// x = (double *)malloc(sizeof(double)*Nx);
	x = dvector(Nx); y = dvector(Ny);
	u_h = dvector(Nx*Ny); oldu_h = dvector(Nx*Ny);
	exc = dvector(Nx*Ny);
	u_d = dvector(Nx*Ny);

	for (j = 0; j < Nx-1; j++) x[j] = 0.0 + (j*h);
	for (i = 0; i < Ny-1; i++) y[i] = 0.0 + (i*h);

	zero_matrix(u_h, Nx, Ny);
	zero_matrix(oldu_h, Nx, Ny);
	zero_matrix(exc, Nx, Ny);

	// initial
	initialize(u_h, x, y, Nx, Ny);
	initialize(oldu_h, x, y, Nx, Ny);

	// allocate temperature arrays on device
	dev_matrix<double> ud(Nx, Ny); ud.set(u_h, Nx, Ny);
	dev_matrix<double> oldud(Nx, Ny); oldud.set(u_h, Nx, Ny);
	dev_matrix<double> tmp_d(Nx, Ny);

	// set threads and blocks
	numBlocks = dim3(iDivUp(Nx,BLOCK_SIZE_X), iDivUp(Ny,BLOCK_SIZE_Y));
	threadsPerBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// cpu loop
	printf("CPU start!\n");
	clock_h = double(clock()) / CLOCKS_PER_SEC;
	for (iter = 0; iter < Nt; iter++) {
		heat2d_cpu(Nx, Ny, alpha, oldu_h, u_h);
        tmp_h = u_h;
        u_h = oldu_h;
        oldu_h = tmp_h;
	}
	clock_h = double(clock()) / CLOCKS_PER_SEC - clock_h;
	printf("CPU end!\n");

	// gpu loop
	printf("GPU start!\n");
	clock_d = double(clock()) / CLOCKS_PER_SEC;
	for (iter = 0; iter < Nt; iter++) {
		heat2d_gpu<<<numBlocks, threadsPerBlock>>>(Nx, Ny, alpha, oldud.getData(), ud.getData());
        tmp_d = ud;
        ud = oldud;
        oldud = tmp_d;
	} 
	hipDeviceSynchronize();
	clock_d = double(clock()) / CLOCKS_PER_SEC - clock_d;
	printf("GPU end!\n");

	// copy temperature array from device to host
	oldud.get(&u_d[0], Nx, Ny);

	// Exact value
	heat2d_exc(exc, x, y, T, Nx, Ny);

	//RMSE
	for (i = 0; i < Nx*Ny; i++) {			
		errGPU = errGPU + ( (u_d[i]-exc[i])*(u_d[i]-exc[i]) );
		errCPU = errCPU + ( (oldu_h[i]-exc[i])*(oldu_h[i]-exc[i]) );
	}
	errGPU = sqrt(errGPU / (Nx*Ny));
	errCPU = sqrt(errCPU / (Nx*Ny));
	printf("\n");
	printf("RMSE (CPU) : %.12f\n", errCPU);
	printf("RMSE (GPU) : %.12f\n", errGPU);
	printf("CPU time = %.3fms\n",clock_h*1e3);
	printf("GPU time = %.3fms\n",clock_d*1e3);
	printf("CPU time / GPU time : %.2f\n", clock_h/clock_d);

	printf("\n");
	printf("Printing...\n");
	fp = fopen("host_out.dat", "w");
	print_mat(fp, oldu_h, Nx, Ny);
	
	fp = fopen("dev_out.dat", "w");
	print_mat(fp, u_d, Nx, Ny);
	
	fp = fopen("exact.dat", "w");
	print_mat(fp, exc, Nx, Ny);


	oldud.~dev_matrix(); ud.~dev_matrix();
	free(x); free(y); free(u_h); free(oldu_h);
	free(exc); free(u_d);

	return 0;
}


